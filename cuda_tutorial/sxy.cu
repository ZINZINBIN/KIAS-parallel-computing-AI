// written by Jongsoo Kim
// revision history:  
// 20210711 JK added #progma openACC compiler directive
// 
// nvcc -std=c++11 -o sxy sxy.cu 


#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>  // for measuring execution time
using namespace std::chrono;

__global__ void sxy(const float *x, const float *y, float *z, const unsigned int n) 
{ 
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // index for arrays 
    while (i < n) 
    {
       z[i] = x[i] + y[i];
       i += blockDim.x * gridDim.x;
    }
}
     
int main (void) 
{
    const unsigned int n = 1 << 20;  // power ofint 2

    // allocate the memory on host 
    float *x = new float[n];
    float *y = new float[n];
    float *z = new float[n];

    // allocate the memory on device 
    float *dev_x, *dev_y, *dev_z;
    hipMalloc( &dev_x, n*sizeof(float) );
    hipMalloc( &dev_y, n*sizeof(float) );
    hipMalloc( &dev_z, n*sizeof(float) );

    for (int i=0; i<n; i++) 
    {
        x[i] = static_cast<float>(i);
        y[i] = static_cast<float>(i);
    }

    // starting time
    auto start = high_resolution_clock::now();

    // copy the host arrays 'x' and 'y' to device 
    hipMemcpy ( dev_x, x, n*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy ( dev_y, y, n*sizeof(float), hipMemcpyHostToDevice );

    sxy<<<256,256>>>(dev_x, dev_y, dev_z, n);

    // copy the device array 'z' to host 
    hipMemcpy ( z, dev_z, n*sizeof(float), hipMemcpyDeviceToHost );

    // end time
    auto end = high_resolution_clock::now();

     // the required to call saxpy
    auto duration = duration_cast<microseconds>(end-start);
    std::cout << "time taken by sxy function  " << duration.count() << "  microseconds" << std::endl;
    std::cout << n/duration.count() << "  MFLOPS" << std::endl;

    // write the results of the first and last vector sums
    std::cout << "first element: " << x[0] << "+" << y[0] << "=" << z[0] << std::endl;
    std::cout << "last element: " << x[n-1] << "+" << y[n-1] << "=" << z[n-1] << std::endl;

    // free the memory allocated on host 
    delete [] x;
    delete [] y;
    delete [] z;

    // free the memory allocated on device 
    hipFree (x);
    hipFree (y);
    hipFree (z);

    return 0;
}
