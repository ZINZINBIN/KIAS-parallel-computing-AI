#define NX 64
#define NY 64
#define NZ 128

hipfftHandle plan;
hipfftComplex *data1, *data2;
hipMalloc((void**)&data1, sizeof(hipfftComplex)*NX*NY*NZ);
hipMalloc((void**)&data2, sizeof(hipfftComplex)*NX*NY*NZ);
/* Create a 3D FFT plan. */
hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C);

/* Transform the first signal in place. */
hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

/* Transform the second signal using the same plan. */
hipfftExecC2C(plan, data2, data2, HIPFFT_FORWARD);

/* Destroy the cuFFT plan. */
hipfftDestroy(plan);
hipFree(data1); hipFree(data2);
