// DFT of a symmetric rectangular function
// written by Jongsoo Kim, 2016-06-18
// nvcc -l cufft cufft_c2c_rect.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

#define N 64 

int main(int argc, char **argv)
{
    FILE *in_file, *out_file;
    // size of memory space for input and output data
    int mem_size = sizeof(hipfftComplex) * N;

    // define data in host memory
    hipfftComplex *h_data = (hipfftComplex *)malloc(mem_size);

    // Since the in-place transform will be used, only one array for the fft    
    // is enough. But it is usually more efficient to use the out-of-place transform. 
    hipfftComplex *d_data;
    hipMalloc( (void **)&d_data,mem_size );

    // initialization of input data
    for (int i = 0; i < N; ++i)
    {
        h_data[i].x = 0.0f;  // real part
        h_data[i].y = 0.0f;  // imaginary part
    }

    // data are symmetric with respect to i=0
    // set the unit values for the first six points 
//    h_data[0].x = 0.5f;
    for (int i = 0; i < N/16+1; ++i) h_data[i].x = 1.0f;

    // set the unit values for the last five points 
    for (int i = 15*N/16; i < N; ++i) h_data[i].x = 1.0f;

    in_file = fopen("cufft_c2c_rect_in.dat","w+");
    for (int i=0; i<N; i++) fprintf(in_file,"%d %e %e\n", i, h_data[i].x, h_data[i].y);
    fclose(in_file);
 
    hipMemcpy(d_data, h_data, mem_size, hipMemcpyHostToDevice);

    hipfftHandle plan; 
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipMemcpy(h_data, d_data, mem_size, hipMemcpyDeviceToHost);

    out_file = fopen("cufft_c2c_rect_out.dat","w+");
    for (int i=0; i<N; i++) fprintf(out_file,"%d %e %e\n", i, h_data[i].x, h_data[i].y);
    fclose(out_file);

    hipfftDestroy(plan); 
    free(h_data);
    hipFree(d_data);
}
