// written by Jongsoo Kim
// last modification: June 24, 2016
// nvcc -l cufft cufft_r2c_sampled_data.cu
// reference: http://holometer.fnal.gov/GH_FFT.pdf


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <time.h>    // for the clock() function
#define TWOPI 6.28318530717959 

__global__ void power_spectrum(hipfftComplex *, hipfftReal *, size_t);

const int N=4096;
const int BATCH_SIZE = 1;
const int OUTPUT_SIZE = N/2+1;
const float fs = 10000.0;              /* sampling frequency [Hz] */

void samples_gen (float *samples)
{
    float f1 = 1234.0;             /* first signal frequency [Hz] */
    float amp1 = 2.82842712474619; /* 2 Vrms */
    float f2 = 2500.2157;          /* second signal frequency [Hz] */
    float amp2 = 1.0;              /* 0.707 Vrms */
    float ulsb = 1.0e-3;           /* Value of 1 LSB in Volt */
    float t, u;

    for (size_t i = 0; i < N*BATCH_SIZE; i++)
    {
        t = (float) i / fs;
        u = amp1 * sin (TWOPI * f1 * t) + amp2 * sin (TWOPI * f2 * t);
        samples[i] = floor (u / ulsb + 0.5) * ulsb; /* Rounding */
    }
}

int main (int argc, char *argv[])
{
    FILE *out_file;

    float fres = fs/(float) N;

    hipfftReal *h_in;
    h_in = (hipfftReal *)malloc(sizeof(hipfftReal) * N * BATCH_SIZE);
    hipfftReal *h_ps;
    h_ps = (hipfftReal *)malloc(sizeof(hipfftReal) * OUTPUT_SIZE);

    // generate input data
    samples_gen (h_in);

    double etime = (double) clock(); // start to measure the elapsed time

    hipfftReal *d_in;  
    hipfftComplex *d_out; 
    hipfftReal *d_ps; 

    hipMalloc( (void **)&d_in, sizeof(hipfftReal) * N * BATCH_SIZE);
    hipMalloc( (void **)&d_out,sizeof(hipfftComplex) * OUTPUT_SIZE * BATCH_SIZE);
    hipMalloc( (void **)&d_ps, sizeof(hipfftReal) * OUTPUT_SIZE );
    // initialize d_ps array
    hipMemset( d_ps, 0.0f, sizeof(hipfftReal) * OUTPUT_SIZE );

    hipMemcpy(d_in, h_in, sizeof(hipfftReal) * N * BATCH_SIZE, hipMemcpyHostToDevice);

    // make a plan for fft
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_R2C, BATCH_SIZE);
    hipfftExecR2C(plan, d_in, d_out);

    // calculation of power spectrum
    int numThreads = 128;
    int numGrids = (OUTPUT_SIZE*BATCH_SIZE + numThreads -1)/numThreads;
    printf("%d %d\n", numThreads, numGrids);
    power_spectrum<<<numGrids,numThreads>>>(d_out,d_ps,OUTPUT_SIZE);
    hipMemcpy(h_ps, d_ps, sizeof(hipfftReal)*OUTPUT_SIZE, hipMemcpyDeviceToHost);

    etime = ((double) clock() - etime)/(double)CLOCKS_PER_SEC ;
    printf("elapsed time in seconds = %e\n",etime);
    double num_ops = BATCH_SIZE*5.0*(double) N * log10(N);
    printf("number of floating point operations = %e\n",num_ops);
    printf("%e Gflops\n",num_ops/etime/1.e9);

    out_file = fopen("cufft_r2c_sampled_data_out.dat","w+");
    for (int i=0;i<N/2+1;++i) fprintf(out_file,"%e %e \n", i*fres/1000.0, sqrt(2.0*h_ps[i]/(hipfftReal)BATCH_SIZE/fs));

    hipfftDestroy(plan);
    free(h_in); free(h_ps);
    hipFree(d_in); hipFree(d_out); hipFree(d_ps);

    return 0;
}

__global__ void power_spectrum(hipfftComplex *c, hipfftReal *a, size_t OUTPUT_SIZE)
{
    const size_t numThreads = blockDim.x * gridDim.x;
    const size_t threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = threadID; i < OUTPUT_SIZE*BATCH_SIZE ; i += numThreads)
    {
        size_t j = i % OUTPUT_SIZE;
        atomicAdd( &a[j], c[i].x*c[i].x+c[i].y*c[i].y);
    }
}
