// DFT of a symmetric rectangular function
// written by Jongsoo Kim, 2016-06-18
// nvcc -l cufft cufft_r2c_rect.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

#define N 64 

int main(int argc, char **argv)
{
    FILE *in_file, *out_file;

    // define in and out pointers in host memory
    hipfftReal    *h_in = (hipfftReal *)malloc(sizeof(hipfftReal) * N);
    hipfftComplex *h_out = (hipfftComplex *)malloc(sizeof(hipfftComplex) * (N/2+1));

    // define in and out pointers in device memory
    hipfftReal    *d_in;
    hipfftComplex *d_out;
    hipMalloc( (void **)&d_in, sizeof(hipfftReal) * N);
    hipMalloc( (void **)&d_out,sizeof(hipfftComplex) * (N/2+1));

    // initialization of input data
    for (int i = 0; i < N; ++i)
    {
        h_in[i] = 0.0f;  // real part
    }

    // data are symmetric with respect to i=0
    // set the unit values for the first six points
    for (int i = 0; i < 6; ++i) h_in[i] = 1.0f;

    // set the unit values for the last five points
    for (int i = N-5; i < N; ++i) h_in[i] = 1.0f;

    in_file = fopen("cufft_r2c_rect_in.dat","w+");
    for (int i=0; i<N; i++) fprintf(in_file,"%d %e \n", i, h_in[i]);
    fclose(in_file);

    hipMemcpy(d_in, h_in, sizeof(hipfftReal)*N, hipMemcpyHostToDevice);

    hipfftHandle plan; 
    hipfftPlan1d(&plan, N, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, d_in, d_out);
    hipMemcpy(h_out, d_out, sizeof(hipfftComplex) * (N/2+1), hipMemcpyDeviceToHost);

    out_file = fopen("cufft_r2c_rect_out.dat","w+");
    for (int i=0; i<N/2+1; i++) fprintf(out_file,"%d %e %e\n", i, h_out[i].x, h_out[i].y);
    fclose(out_file);

    hipfftDestroy(plan); 
    free(h_in); free(h_out);
    hipFree(d_in); hipFree(d_out);
}
