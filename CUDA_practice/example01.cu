#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void kernel(void){
    printf("kernel test\n");
}

int main(int argc, char *argv[]){

    kernel <<<1,5>>>();
    hipDeviceSynchronize();

    // cudaDeviceReset();
    return 0;
}