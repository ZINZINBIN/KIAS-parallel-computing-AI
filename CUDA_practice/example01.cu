#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void kernel(void){
    printf("kernel test\n");
}

int main(int argc, char *argv[]){

    kernel <<<1,5>>>();
    hipDeviceReset();
    return 0;
}